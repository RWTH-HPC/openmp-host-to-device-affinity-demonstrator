#include "hip/hip_runtime.h"
#include <stdlib.h>

#include "kernel.hpp"
#include "../util/define.hpp"

#if (USE_OMP_TARGET == 0)

using namespace kernel;

//static std::vector<hipStream_t> streams;

__global__ void matrix_mutliply(
        double const *a, double const *b, double *c, int const n) {
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    float tmpSum = 0;

    if (row < n && col < n) {
        // each thread computes one element of the block sub-matrix
        for (int i = 0; i < n; i++) {
            tmpSum += a[row * n + i] * b[i * n + col];
        }
        c[row * n + col] = tmpSum;
    }
}

MatrixMultiplyCUDA::MatrixMultiplyCUDA(int device, int num_streams) : MatrixMultiplyDevice(device) {
    hipSetDevice(device);
    streams = (hipStream_t*)malloc(sizeof(hipStream_t) * num_streams);
    for (int i = 0; i < num_streams; i++) {
        hipStream_t current;
        hipStreamCreate(&current);
        streams[i] = current;
    }
}

MatrixMultiplyCUDA::~MatrixMultiplyCUDA() {
    free(streams);
}

void MatrixMultiplyCUDA::execute(
        double const *a, double const *b, double *c, int const n) const {
    hipSetDevice(device);
#if (COMPUTE == 1)
    dim3 threads_per_block(n, n);
    dim3 blocks_per_grid(1, 1);
    if (n*n > 1024){
        threads_per_block.x = 32;
        threads_per_block.y = 32;
        blocks_per_grid.x = (n + threads_per_block.x - 1) / threads_per_block.x;
        blocks_per_grid.y = (n + threads_per_block.y - 1) / threads_per_block.y;
    }
#endif
    double *d_a;
    double *d_b;
    double *d_c;

    int size = sizeof(double) * n*n;

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);


    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

#if (COMPUTE == 1)
    matrix_mutliply<<<blocks_per_grid,threads_per_block, 0>>>(d_a, d_b, d_c, n);
#endif

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

}

void MatrixMultiplyCUDA::executeAsync(
        double const *a, double const *b, double *c, int const n, int const stream_id) const {

    hipSetDevice(device);
#if (COMPUTE == 1)
    dim3 threads_per_block(n, n);
    dim3 blocks_per_grid(1, 1);
    if (n*n > 1024){
        threads_per_block.x = 32;
        threads_per_block.y = 32;
        blocks_per_grid.x = (n + threads_per_block.x - 1) / threads_per_block.x;
        blocks_per_grid.y = (n + threads_per_block.y - 1) / threads_per_block.y;
    }
#endif
    double *d_a;
    double *d_b;
    double *d_c;

    int size = sizeof(double) * n*n;

    hipStream_t stream = streams[stream_id];

    hipMallocAsync((void **)&d_a, size, stream);
    hipMallocAsync((void **)&d_b, size, stream);
    hipMallocAsync((void **)&d_c, size, stream);


    hipMemcpyAsync(d_a, a, size, hipMemcpyDefault, stream);
    hipMemcpyAsync(d_b, b, size, hipMemcpyDefault, stream);

#if (COMPUTE == 1)
    matrix_mutliply<<<blocks_per_grid, threads_per_block, 0, stream>>>(d_a, d_b, d_c, n);
#endif

    hipMemcpyAsync(c, d_c, size, hipMemcpyDefault, stream);

    hipFreeAsync(d_a, stream);
    hipFreeAsync(d_b, stream);
    hipFreeAsync(d_c, stream);
}

void MatrixMultiplyCUDA::syncronizeStream(int const stream_id) const {
    hipStreamSynchronize(streams[stream_id]);
    hipStreamDestroy(streams[stream_id]);
}
#endif // USE_OMP_TARGET
