#include "hip/hip_runtime.h"
#include <stdlib.h>

#include "../util/define.hpp"
#include "kernel.hpp"

#if (USE_OMP_TARGET == 0)

using namespace kernel;

// static std::vector<hipStream_t> streams;

__global__ void matrix_mutliply(double const *a, double const *b, double *c, size_t const n)
{
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;

    float tmpSum = 0;

    if (row < n && col < n)
    {
        // each thread computes one element of the block sub-matrix
        for (size_t i = 0; i < n; i++)
        {
            tmpSum += a[row * n + i] * b[i * n + col];
        }
        c[row * n + col] = tmpSum;
    }
}

MatrixMultiplyCUDA::MatrixMultiplyCUDA(int device, int num_streams) : MatrixMultiplyDevice(device)
{
    gpuErrChk(hipSetDevice(device));
    streams = (hipStream_t *)malloc(sizeof(hipStream_t) * num_streams);
    for (int i = 0; i < num_streams; i++)
    {
        hipStream_t current;
        gpuErrChk(hipStreamCreate(&current));
        streams[i] = current;
    }
}

MatrixMultiplyCUDA::~MatrixMultiplyCUDA()
{
    free(streams);
}

void MatrixMultiplyCUDA::execute(double const *a, double const *b, double *c, size_t const n) const
{
    gpuErrChk(hipSetDevice(device));
#if (COMPUTE == 1)
    dim3 threads_per_block(n, n);
    dim3 blocks_per_grid(1, 1);
    if (n * n > 1024)
    {
        threads_per_block.x = 32;
        threads_per_block.y = 32;
        blocks_per_grid.x = (n + threads_per_block.x - 1) / threads_per_block.x;
        blocks_per_grid.y = (n + threads_per_block.y - 1) / threads_per_block.y;
    }
#endif // COMPUTE == 1
    double *d_a;
    double *d_b;
    double *d_c;

    size_t size = sizeof(double) * n * n;

    gpuErrChk(hipMalloc((void **)&d_a, size));
    gpuErrChk(hipMalloc((void **)&d_b, size));
    gpuErrChk(hipMalloc((void **)&d_c, size));

    gpuErrChk(hipMemcpy(d_a, a, size, hipMemcpyHostToDevice));
    gpuErrChk(hipMemcpy(d_b, b, size, hipMemcpyHostToDevice));

#if (COMPUTE == 1)
    matrix_mutliply<<<blocks_per_grid, threads_per_block, 0>>>(d_a, d_b, d_c, n);
#endif // COMPUTE == 1

    gpuErrChk(hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost));

    gpuErrChk(hipFree(d_a));
    gpuErrChk(hipFree(d_b));
    gpuErrChk(hipFree(d_c));
}

void MatrixMultiplyCUDA::executeAsync(double const *a, double const *b, double *c, size_t const n,
                                      int const stream_id) const
{

    gpuErrChk(hipSetDevice(device));
#if (COMPUTE == 1)
    dim3 threads_per_block(n, n);
    dim3 blocks_per_grid(1, 1);
    if (n * n > 1024)
    {
        threads_per_block.x = 32;
        threads_per_block.y = 32;
        blocks_per_grid.x = (n + threads_per_block.x - 1) / threads_per_block.x;
        blocks_per_grid.y = (n + threads_per_block.y - 1) / threads_per_block.y;
    }
#endif // COMPUTE == 1
    double *d_a;
    double *d_b;
    double *d_c;

    size_t size = sizeof(double) * n * n;

    hipStream_t stream = streams[stream_id];

    gpuErrChk(hipMallocAsync((void **)&d_a, size, stream));
    gpuErrChk(hipMallocAsync((void **)&d_b, size, stream));
    gpuErrChk(hipMallocAsync((void **)&d_c, size, stream));

    gpuErrChk(hipMemcpyAsync(d_a, a, size, hipMemcpyDefault, stream));
    gpuErrChk(hipMemcpyAsync(d_b, b, size, hipMemcpyDefault, stream));

#if (COMPUTE == 1)
    matrix_mutliply<<<blocks_per_grid, threads_per_block, 0, stream>>>(d_a, d_b, d_c, n);
#endif // COMPUTE == 1

    gpuErrChk(hipMemcpyAsync(c, d_c, size, hipMemcpyDefault, stream));

    gpuErrChk(hipFreeAsync(d_a, stream));
    gpuErrChk(hipFreeAsync(d_b, stream));
    gpuErrChk(hipFreeAsync(d_c, stream));
}

void MatrixMultiplyCUDA::syncronizeStream(int const stream_id) const
{
    gpuErrChk(hipStreamSynchronize(streams[stream_id]));
    gpuErrChk(hipStreamDestroy(streams[stream_id]));
}
#endif // USE_OMP_TARGET
