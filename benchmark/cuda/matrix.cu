#include "hip/hip_runtime.h"
#include "../util/define.hpp"

#include "matrix.cuh"
#include <cstddef>
#include <vector>
#include <omp.h>

static std::vector<hipStream_t> streams;

__global__ void matrix_mutliply(
        const double *a, const double *b, double *c, const unsigned int n) {
    //__shared__ double a_block[BLOCK_SIZE*BLOCK_SIZE];
    //__shared__ double b_block[BLOCK_SIZE*BLOCK_SIZE];

    //int tasks_per_thread = (BLOCK_SIZE*BLOCK_SIZE + blockDim.x*blockDim.y - 1)/(blockDim.x*blockDim.y);

    //int *idx = new int[tasks_per_thread];
    //int *idy = new int[tasks_per_thread];
    //double *tmp = new double[tasks_per_thread];

    //for (int task = 0; task < tasks_per_thread; task++) {
    //    int tx = ((threadIdx.x + threadIdx.y * BLOCK_SIZE) * tasks_per_thread + task) % BLOCK_SIZE;
    //    int ty = ((threadIdx.x + threadIdx.y * BLOCK_SIZE) * tasks_per_thread + task) / BLOCK_SIZE;

    //    idx[task] = tx + BLOCK_SIZE * blockIdx.x;
    //    idy[task] = ty + BLOCK_SIZE * blockIdx.y;

    //    tmp[task] = 0;
    //}


    //for (int i = 0; i < (n+BLOCK_SIZE-1)/BLOCK_SIZE; i++) {
    //    for (int task = 0; task < tasks_per_thread; task++) {
    //        int tx = ((threadIdx.x + threadIdx.y * BLOCK_SIZE) * tasks_per_thread + task) % BLOCK_SIZE;
    //        int ty = ((threadIdx.x + threadIdx.y * BLOCK_SIZE) * tasks_per_thread + task) / BLOCK_SIZE;

    //        if (i*BLOCK_SIZE + tx < n)
    //            a_block[tx + ty * BLOCK_SIZE] = a[idy[task]*n + (i*BLOCK_SIZE + tx)];
    //        else
    //            a_block[tx + ty * BLOCK_SIZE] = 0;

    //        if (i*BLOCK_SIZE + ty < n)
    //            b_block[tx + ty * BLOCK_SIZE] = b[(i*BLOCK_SIZE + ty)*n + (idx[task])];
    //        else
    //            b_block[tx + ty * BLOCK_SIZE] = 0;
    //    }

    //    __syncthreads();

    //    for (int task = 0; task < tasks_per_thread; task++) {
    //        int tx = ((threadIdx.x + threadIdx.y * BLOCK_SIZE) * tasks_per_thread + task) % BLOCK_SIZE;

    //        for (int j = 0; j < BLOCK_SIZE; j++) {
    //            tmp[task] += a_block[tx * BLOCK_SIZE + j] * b_block[j * BLOCK_SIZE + tx];
    //        }
    //    }
    //    __syncthreads();
    //}
    //for (int task = 0; task < tasks_per_thread; task++) {
    //    if (idx[task] < n && idy[task] < n) {
    //        c[idy[task]*n+idx[task]] = tmp[task];
    //    }
    //}

    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    float tmpSum = 0;

    if (row < n && col < n) {
        // each thread computes one element of the block sub-matrix
        for (int i = 0; i < n; i++) {
            tmpSum += a[row * n + i] * b[i * n + col];
        }
        c[row * n + col] = tmpSum;
    }

    //int64_t cycles = 0;
    //int64_t start = clock64();
    //while(cycles < 1480500 * 100) {
    //    cycles = clock64() - start;
    //}


}

void kernel::execute_matrix_multiply_kernel(const double *a, 
        const double *b, 
        double *c, 
        const unsigned int n,
        const int device) {

    hipSetDevice(device);
#if (COMPUTE == 1)
    dim3 threads_per_block(n, n);
    dim3 blocks_per_grid(1, 1);
    if (n*n > 1024){
        threads_per_block.x = 32;
        threads_per_block.y = 32;
        blocks_per_grid.x = (n + threads_per_block.x - 1) / threads_per_block.x;
        blocks_per_grid.y = (n + threads_per_block.y - 1) / threads_per_block.y;
    }
#endif
    double *d_a;
    double *d_b;
    double *d_c;

    int size = sizeof(double) * n*n;

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);


    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

#if (COMPUTE == 1)
    matrix_mutliply<<<blocks_per_grid,threads_per_block, 0>>>(d_a, d_b, d_c, n);
#endif

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

}

void kernel::execute_matrix_multiply_kernel_async(const double *a, 
        const double *b, 
        double *c, 
        const unsigned int n,
        const int stream_id,
        const int device) {

    hipSetDevice(device);
#if (COMPUTE == 1)
    dim3 threads_per_block(n, n);
    dim3 blocks_per_grid(1, 1);
    if (n*n > 1024){
        threads_per_block.x = 32;
        threads_per_block.y = 32;
        blocks_per_grid.x = (n + threads_per_block.x - 1) / threads_per_block.x;
        blocks_per_grid.y = (n + threads_per_block.y - 1) / threads_per_block.y;
    }
#endif
    double *d_a;
    double *d_b;
    double *d_c;

    int size = sizeof(double) * n*n;

    hipStream_t stream = streams[stream_id];

    hipMallocAsync((void **)&d_a, size, stream);
    hipMallocAsync((void **)&d_b, size, stream);
    hipMallocAsync((void **)&d_c, size, stream);


    hipMemcpyAsync(d_a, a, size, hipMemcpyDefault, stream);
    hipMemcpyAsync(d_b, b, size, hipMemcpyDefault, stream);

#if (COMPUTE == 1)
    matrix_mutliply<<<blocks_per_grid, threads_per_block, 0, stream>>>(d_a, d_b, d_c, n);
#endif

    hipMemcpyAsync(c, d_c, size, hipMemcpyDefault, stream);

    hipFreeAsync(d_a, stream);
    hipFreeAsync(d_b, stream);
    hipFreeAsync(d_c, stream);
}

void kernel::initStreams(const int num_streams) {
    streams = std::vector<hipStream_t>(num_streams);
}

void kernel::createStream(const int stream_id, const int device) {
    hipSetDevice(device);

    hipStream_t cur;
    hipStreamCreate(&cur);
    streams[stream_id] = cur;
}

void kernel::syncronizeStream(const int stream_id) {
    hipStreamSynchronize(streams[stream_id]);
    hipStreamDestroy(streams[stream_id]);
}

void kernel::pin(void *data, size_t size, bool readonly, const int device) {
    hipSetDevice(device);
    if (readonly)
        hipHostRegister(data, size, hipHostRegisterReadOnly);
    else
        hipHostRegister(data, size, hipHostRegisterDefault);
}

void kernel::unpin(void *data) {
    hipHostUnregister(data);
}

void *kernel::hostPinnedMalloc(size_t size, const int device) {
    void *ptr;

#if (UNIFIED_MEMORY == 0)
    hipSetDevice(device);
    hipHostAlloc(&ptr, size, hipHostMallocDefault);
#elif (UNIFIED_MEMORY == 1)
    hipHostAlloc(&ptr, size, hipHostMallocPortable);
#endif
    return ptr;
}

void kernel::hostPinnedFree(void *data) {
    hipHostFree(data);
}
