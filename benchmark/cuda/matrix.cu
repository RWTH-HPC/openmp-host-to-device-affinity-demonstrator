#include "hip/hip_runtime.h"
#include "../util/define.hpp"

#include "matrix.cuh"
#include <cstddef>
#include <vector>
#include <omp.h>


__global__ void matrix_mutliply(
        const double *a, const double *b, double *c, const unsigned int n) {
    __shared__ double a_block[BLOCK_SIZE*BLOCK_SIZE];
    __shared__ double b_block[BLOCK_SIZE*BLOCK_SIZE];

    //index in the total matrix for which block entry the thread is responsible for
    int idx = threadIdx.x + BLOCK_SIZE * blockIdx.x;
    int idy = threadIdx.y + BLOCK_SIZE * blockIdx.y;

    if (idx < n && idy < n) {
        double tmp = 0;
        for (int i = 0; i < n/BLOCK_SIZE; i++) {
            a_block[threadIdx.x + threadIdx.y * BLOCK_SIZE] = a[idy*n + (i*BLOCK_SIZE + threadIdx.x)];
            b_block[threadIdx.x + threadIdx.y * BLOCK_SIZE] = b[(i*BLOCK_SIZE + threadIdx.y)*n + (idx)];
            __syncthreads();

            for (int j = 0; j < BLOCK_SIZE; j++) {
                tmp += a_block[threadIdx.y * BLOCK_SIZE + j] * b_block[j * BLOCK_SIZE + threadIdx.x];
            }
            __syncthreads();
        }
        c[idy*n+idx] = tmp;
    }
}

void kernel::execute_matrix_multiply_kernel(const double *a, 
        const double *b, 
        double *c, 
        const unsigned int n,
        const int device) {

    hipSetDevice(device);
#if (COMPUTE == 1)
    dim3 threads_per_block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks((n+BLOCK_SIZE-1)/BLOCK_SIZE, (n+BLOCK_SIZE-1)/BLOCK_SIZE);
#endif
    double *d_a;
    double *d_b;
    double *d_c;

    int size = sizeof(double) * n*n;

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);


    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

#if (COMPUTE == 1)
    matrix_mutliply<<<blocks,threads_per_block, 0>>>(d_a, d_b, d_c, n);
#endif

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

}

void kernel::syncronize(const int device) {
    hipSetDevice(device);
    hipDeviceSynchronize();
}

void kernel::pin(void *data, size_t size) {
    hipHostRegister(data, size, hipHostRegisterDefault);
}

void kernel::unpin(void *data) {
    hipHostUnregister(data);
}
