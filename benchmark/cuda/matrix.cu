#include "hip/hip_runtime.h"
#include "../util/define.hpp"

#include "matrix.cuh"
#include <cstddef>
#include <vector>
#include <omp.h>


__global__ void matrix_mutliply(
        const double *a, const double *b, double *c, const unsigned int n) {
    __shared__ double a_block[BLOCK_SIZE*BLOCK_SIZE];
    __shared__ double b_block[BLOCK_SIZE*BLOCK_SIZE];

    //index in the total matrix for which block entry the thread is responsible for
    int idx = threadIdx.x + BLOCK_SIZE * blockIdx.x;
    int idy = threadIdx.y + BLOCK_SIZE * blockIdx.y;

    if (idx < n && idy < n) {
        double tmp = 0;
        for (int i = 0; i < n/BLOCK_SIZE; i++) {
            a_block[threadIdx.x + threadIdx.y * BLOCK_SIZE] = a[idy*n + (i*BLOCK_SIZE + threadIdx.x)];
            b_block[threadIdx.x + threadIdx.y * BLOCK_SIZE] = b[(i*BLOCK_SIZE + threadIdx.y)*n + (idx)];
            __syncthreads();

            for (int j = 0; j < BLOCK_SIZE; j++) {
                tmp += a_block[threadIdx.y * BLOCK_SIZE + j] * b_block[j * BLOCK_SIZE + threadIdx.x];
            }
            __syncthreads();
        }
        c[idy*n+idx] = tmp;
    }
}

void kernel::execute_matrix_multiply_kernel(const double *a, 
        const double *b, 
        double *c, 
        const unsigned int n,
        const int device) {

    hipSetDevice(device);
#if (COMPUTE == 1)
    dim3 threads_per_block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks((n+BLOCK_SIZE-1)/BLOCK_SIZE, (n+BLOCK_SIZE-1)/BLOCK_SIZE);
#endif
    double *d_a;
    double *d_b;
    double *d_c;

    int size = sizeof(double) * n*n;

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);


    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

#if (COMPUTE == 1)
    matrix_mutliply<<<blocks,threads_per_block, 0>>>(d_a, d_b, d_c, n);
#endif

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

}

void kernel::execute_matrix_multiply_kernel_async(const double *a, 
        const double *b, 
        double *c, 
        const unsigned int n,
        const int device) {

    hipSetDevice(device);
#if (COMPUTE == 1)
    dim3 threads_per_block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks((n+BLOCK_SIZE-1)/BLOCK_SIZE, (n+BLOCK_SIZE-1)/BLOCK_SIZE);
#endif
    double *d_a;
    double *d_b;
    double *d_c;

    hipStream_t stream;
    hipStreamCreate(&stream);

    int size = sizeof(double) * n*n;

    hipMallocAsync((void **)&d_a, size, stream);
    hipMallocAsync((void **)&d_b, size, stream);
    hipMallocAsync((void **)&d_c, size, stream);


    hipMemcpyAsync(d_a, a, size, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_b, b, size, hipMemcpyHostToDevice, stream);

#if (COMPUTE == 1)
    matrix_mutliply<<<blocks,threads_per_block, 0, stream>>>(d_a, d_b, d_c, n);
#endif

    hipMemcpyAsync(c, d_c, size, hipMemcpyDeviceToHost, stream);

    hipFreeAsync(d_a, stream);
    hipFreeAsync(d_b, stream);
    hipFreeAsync(d_c, stream);

    hipStreamDestroy(stream);
}

void kernel::syncronize(const int device) {
    hipSetDevice(device);
    hipDeviceSynchronize();
}

void kernel::pin(void *data, size_t size, bool readonly, const int device) {
    hipSetDevice(device);
    if (readonly)
        hipHostRegister(data, size, hipHostRegisterReadOnly);
    else
        hipHostRegister(data, size, hipHostRegisterDefault);
}

void kernel::unpin(void *data) {
    hipHostUnregister(data);
}
