#include "hip/hip_runtime.h"
#include "matrix.cuh"
#include "../util/gpu_distance.hpp"
#include <vector>
#include <omp.h>

__global__ void matrix_mutliply(
        const double *a, const double *b, double *c, const unsigned int n) {

    __shared__ double res;

    if (threadIdx.x == 0)
        res = 0;

    double my_val = 
        a[blockIdx.x * n + threadIdx.x] * //a[bx][tx]
        b[threadIdx.x * n + blockIdx.y];  //b[tx][by]

    __syncthreads();

    for (int i = 0; i < blockDim.x; i++) {
        if (threadIdx.x == i) {
            res += my_val;
        }
        __syncthreads();
    }

    if (threadIdx.x == 0)
        c[blockIdx.x * n + blockIdx.y] = res; //c[bx][by]
}

void kernel::execute_matrix_multiply_kernel(const double *a, 
        const double *b, 
        double *c, 
        const unsigned int n,
        const int device) {

    hipSetDevice(device);
    dim3 blocks(n,n,1);
    dim3 threads(n,1,1);

    double *d_a;
    double *d_b;
    double *d_c;

    int size = sizeof(double) * n*n;

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    matrix_mutliply<<<blocks,threads>>>(d_a, d_b, d_c, n);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

void kernel::syncronize(const int device) {
    hipSetDevice(device);
    hipDeviceSynchronize();
}
