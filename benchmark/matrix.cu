#include "hip/hip_runtime.h"
#include "matrix.cuh"
#include <vector>

__global__ void matrix_mutliply(
        const double *a, const double *b, double *c) {
    __shared__ double res;

    if (threadIdx.x == 0)
        res = 0;

    double my_val = 
        a[blockIdx.x * blockDim.x + threadIdx.x] * //a[bx][tx]
        b[threadIdx.x * blockDim.x + blockIdx.y];  //b[tx][by]

    __syncthreads();

    for (int i = 0; i < blockDim.x; i++) {
        if (threadIdx.x == i) {
            res += my_val;
        }
        __syncthreads();
    }

    if (threadIdx.x == 0)
        c[blockIdx.x * blockDim.x + blockIdx.y] = res; //c[bx][by]
}

void kernel::execute_matrix_multiply_kernel_async(const double *a, 
        const double *b, 
        double *c, 
        const unsigned int n,
        const int device) {

    hipSetDevice(device);
    dim3 blocks(n,n,1);
    dim3 threads(n,1,1);

    double *d_a;
    double *d_b;
    double *d_c;

    int size = sizeof(double) * n*n;

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    matrix_mutliply<<<blocks,threads>>>(d_a, d_b, d_c);
    hipMemcpyAsync(c, d_c, size, hipMemcpyDeviceToHost);

    hipFreeAsync(d_a, 0);
    hipFreeAsync(d_b, 0);
    hipFreeAsync(d_c, 0);
}

void kernel::syncronize(const int device) {
    hipSetDevice(device);
    hipDeviceSynchronize();
}
