
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>

constexpr const int N = 1024;
constexpr const int M = 32;

__global__ void cu_vector_mul(const float *a, const float *b, float *c) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    c[index] = a[index] * b[index];
}

int main() {
    float a[N], b[N], c[N];

    srand(time(NULL));
    for (int i = 0; i < N; i++) {
        a[i] = static_cast<float>(rand())/static_cast<float>(RAND_MAX);
        b[i] = static_cast<float>(rand())/static_cast<float>(RAND_MAX);
        c[i] = 0;
    }

    float *d_a;
    float *d_b;
    float *d_c;


    int size = sizeof(float) * N;

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    cu_vector_mul<<<N/M,M>>>(d_a, d_b, d_c);

    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);


    for (int i = 0; i < N; i++) {
        std::cout << a[i] << " * " << b[i] << " = " << c[i] << std::endl;
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}
